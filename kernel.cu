#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "cuda_util.h"

#define ROWS 32
#define COLS 16

__global__ void Kerneltest(int **da, unsigned int rows, unsigned int cols)
{
    unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;
    if (row < rows && col < cols)
    {
        da[row][col] = row*cols + col;

    }

}

int func()
{
    int **da = NULL;
    int **ha = NULL;
    int *dc = NULL;
    int *hc = NULL;
    int r, c;
    bool is_right=true;

    CUDA_CHECK(hipMalloc((void**)(&da), ROWS*sizeof(int*)));
    CUDA_CHECK(hipMalloc((void**)(&dc), ROWS*COLS*sizeof(int)));
    ha = (int**)malloc(ROWS*sizeof(int*));
    hc = (int*)malloc(ROWS*COLS*sizeof(int));

    for (r = 0; r < ROWS; r++)
    {
        ha[r] = dc + r*COLS;
    }
    CUDA_CHECK(hipMemcpy((void*)(da), (void*)(ha), ROWS*sizeof(int*), hipMemcpyHostToDevice));
    dim3 dimBlock(16,16);
    dim3 dimGrid((COLS+dimBlock.x-1)/(dimBlock.x), (ROWS+dimBlock.y-1)/(dimBlock.y));
    Kerneltest<<<dimGrid, dimBlock>>>(da, ROWS, COLS);
    CUDA_CHECK(hipMemcpy((void*)(hc), (void*)(dc), ROWS*COLS*sizeof(int), hipMemcpyDeviceToHost));

    for (r = 0; r < ROWS; r++)
    {
        for (c = 0; c < COLS; c++)
        {   
            printf("%4d ", hc[r*COLS+c]);
            if (hc[r*COLS+c] != (r*COLS+c))
            {   
                is_right = false;
            }   
        }   
        printf("\n");

    }
    printf("the result is %s!\n", is_right? "right":"false");

    hipFree((void*)da);
    hipFree((void*)dc);
    free(ha);
    free(hc);
    //  getchar();
    return 0;

}

